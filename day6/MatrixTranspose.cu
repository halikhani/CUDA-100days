#include <hip/hip_runtime.h>
#include <iostream>

// define size of the matrix
#define WIDTH 1024
#define HEIGHT 1024


__global__ void transposeMatrix(const float* input, float* output, int width, int height){
    // calcluate row and column of the thread

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // transpose if within bounds
    if (col < width && row < height){
        int inputIndex = row * width + col;
        int outputIndex = col * height + row;
        output[outputIndex] = input[inputIndex];
    }
}

// host function for checking cuda errors
void checkCudaError(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << message << " - CUDA Error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main(){
    int width = WIDTH;
    int height = HEIGHT;

    // allocate host memory
    size_t size = width * height * sizeof(float);
    
    float* h_input = (float*)malloc(size);
    float* h_output = (float*)malloc(size);

    // Initialize the input matrix with some values
    for (int i = 0; i < width * height; i++) {
        h_input[i] = static_cast<float>(i);
    }

    // allocate device memory
    float* d_input, *d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // copy data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    checkCudaError("Failed to copy input data to device");

    // define block and grid sizes
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // launch the kernel
    transposeMatrix<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    checkCudaError("Failed to launch kernel");

    // copy data from device to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    checkCudaError("Failed to copy output data to host");

    // Verify the result
    bool success = true;
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            if (h_output[i * height + j] != h_input[j * width + i]) {
                success = false;
                break;
            }
        }
    }

    std::cout << (success ? "Matrix transposition succeeded!" : "Matrix transposition failed!") << std::endl;

    // free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
    
}